#include "hip/hip_runtime.h"
#include "ConcurrentGPUNeuralNet.hh"

#include <cassert>
#include <stdexcept>
#include <cmath>
#include <iostream>
#include <map>
#include <set>
#include <sstream>
#include <vector>
#include <algorithm>

#include "logging.h"
#include "math.h"
#include <fstream>
#include "Timer.hh"



#define cuda_assert(ans) { cudaAssert((ans), __FILE__, __LINE__); }

inline void cudaAssert(hipError_t code, const char *file, int line, bool abort=true) {
//#ifndef NDEBUG
  if (code != hipSuccess) {
    fprintf(stderr,"cuda_assert: %s %s %d\n", hipGetErrorString(code), file, line);
    if (abort) { throw code; }
  }
//#endif
}

std::vector<float> read_matrix(std::string filename, size_t size)
{
  std::vector<float> vec(size,0);

  std::ifstream input_file(filename);
  if (input_file.is_open())
  {
    size_t i = 0;
    while(!input_file.eof())
    {
      input_file >> vec[i];
      i++;
    }
  }
  return vec;
}


//block (1,1,1)
//grid (1,1,1)
__global__ void device_gemm_at_64x256_b_256x32_inner_1(float* A, float* B, float* C)  {
  if (threadIdx.x == 0  && blockIdx.x == 0)
  {
    for (size_t i = 0; i < 64; i++)
    {
      for (size_t j = 0; j < 32; j++)
      {
        float* matC = C + i*32 + j;
        for (size_t k = 0; k < 256; k++)
        {
          float* matA = A + (i*256 + k);
          float* matB = B + (k*32 + j);
          //printf("i = %d, j = %d, k = %d\n", i, j, k);
          //C[i][j] += A[i][k]*B[k][j];
          matC[0] += matA[0]*matB[0];
        }
      }
    }
  }
}


//block (1,1,1)
//grid (1,1,1)
__global__ void device_gemm_at_64x256_b_256x32_outer_1(float* A, float* B, float* C)  {
  if (threadIdx.x == 0  && blockIdx.x == 0)
  {
    for (size_t k = 0; k < 256; k++)
    {
      for (size_t i = 0; i < 64; i++)
      {
        for (size_t j = 0; j < 32; j++)
        {
          float* matC = C + i*32 + j;
          float* matA = A + (i*256 + k);
          float* matB = B + (k*32 + j);
          //printf("i = %d, j = %d, k = %d\n", i, j, k);
          //C[i][j] += A[i][k]*B[k][j];
          matC[0] += matA[0]*matB[0];
        }
      }
    }
  }
}

// block (32, 32, 1)
// grid (2,1,1)
__global__ void device_gemm_at_64x256_b_256x32_outer_2(float* A, float* B, float* C)  {

  size_t i = threadIdx.x + blockIdx.x*blockDim.x;
  size_t j = threadIdx.y;

  for (size_t k = 0; k < 256; k++)
  {
    // printf("i = %d, j = %d, k = %d\n", i, j, k);
    float* matC = C + i*32 + j;
    float* matA = A + (i*256 + k);
    float* matB = B + (k*32 + j);
    matC[0] += matA[0]*matB[0];
  }
}

using Shape = std::vector<float>;

size_t shapesize(const Shape& shape)
{
  size_t size=1;
  for (auto& dim : shape)
  {
    size *= dim;
  }
  return size;
}

template <typename T>
void measure_perf(T global, dim3 grid, dim3 block, float* A_, float* B_, float* C_, std::vector<float>& C, const std::vector<float>& C_expected, Shape shapeC, size_t num_trials)
{
  for (size_t i = 0; i < num_trials; i++)
  {
    global<<<grid,block>>>(A_,B_,C_);
  }

  double tperformance = 0.0;
  for (size_t i = 0; i < num_trials; i++)
  {
    cuda_assert(hipMemset(C_, 0, shapesize(shapeC)*sizeof(float)));
    Timer teval([&tperformance](long long elapsed) { tperformance+=elapsed; });
    global<<<grid,block>>>(A_,B_,C_);
    cuda_assert(hipMemcpy(C.data(),C_, shapesize(shapeC)*sizeof(float),hipMemcpyDeviceToHost));
  }
  std:: cout << tperformance/num_trials/1.0e6 << " ms\n";
  for (auto i=0u; i<shapesize(shapeC); i++) {
    assert(std::abs(C[i]-C_expected[i]) < 1e-4);
  }

  std::cout << "All values match!\n";
}

void ConcurrentGPUNeuralNet::gemm() {
  Shape shapeA = {64,256};
  Shape shapeAt = {256, 64};
  Shape shapeB = {256,32};
  Shape shapeBt = {32, 256};
  Shape shapeC = {64, 32};

  std::vector<float> A = read_matrix("a.dat", shapesize(shapeA));
  std::vector<float> At = read_matrix("at.dat", shapesize(shapeAt));
  std::vector<float> B = read_matrix("b.dat", shapesize(shapeB));
  std::vector<float> Bt = read_matrix("bt.dat", shapesize(shapeBt));
  std::vector<float> C_expected = read_matrix("c.dat", shapesize(shapeC));
  std::vector<float> C(shapesize(shapeC), 0);

  float* A_ = nullptr;
  float* At_ = nullptr;
  float* B_ = nullptr;
  float* Bt_ = nullptr;
  float* C_ = nullptr;
  cuda_assert(hipMalloc((void**)&A_,shapesize(shapeA)*sizeof(float)));
  cuda_assert(hipMalloc((void**)&At_,shapesize(shapeAt)*sizeof(float)));
  cuda_assert(hipMalloc((void**)&B_,shapesize(shapeB)*sizeof(float)));
  cuda_assert(hipMalloc((void**)&Bt_,shapesize(shapeBt)*sizeof(float)));
  cuda_assert(hipMalloc((void**)&C_,shapesize(shapeC)*sizeof(float)));
  cuda_assert(hipMemcpy(A_,A.data(),shapesize(shapeA)*sizeof(float),hipMemcpyHostToDevice));
  cuda_assert(hipMemcpy(At_,At.data(),shapesize(shapeAt)*sizeof(float),hipMemcpyHostToDevice));
  cuda_assert(hipMemcpy(B_,B.data(),shapesize(shapeB)*sizeof(float),hipMemcpyHostToDevice));
  cuda_assert(hipMemcpy(Bt_,Bt.data(),shapesize(shapeBt)*sizeof(float),hipMemcpyHostToDevice));

  std::cout << "device_gemm_at_64x256_b_256x32_inner_1: \n";
  measure_perf(device_gemm_at_64x256_b_256x32_inner_1, dim3{1,1,1}, dim3{1,1,1}, A_,B_,C_,C,C_expected,shapeC,10);
  std::cout << "device_gemm_at_64x256_b_256x32_outer_1: \n";
  measure_perf(device_gemm_at_64x256_b_256x32_outer_1, dim3{1,1,1}, dim3{1,1,1}, A_,B_,C_,C,C_expected,shapeC,10);
  std::cout << "device_gemm_at_64x256_b_256x32_outer_2: \n";
  measure_perf(device_gemm_at_64x256_b_256x32_outer_2, dim3{2,1,1}, dim3{32,32,1}, A_,B_,C_,C,C_expected,shapeC,10);

  cuda_assert(hipFree(A_));
  cuda_assert(hipFree(At_));
  cuda_assert(hipFree(B_));
  cuda_assert(hipFree(Bt_));
  cuda_assert(hipFree(C_));
}


ConcurrentGPUNeuralNet::~ConcurrentGPUNeuralNet() {
  if (node_) { cuda_assert(hipFree(node_)); }
  if (origin_) { cuda_assert(hipFree(origin_)); }
  if (dest_) { cuda_assert(hipFree(dest_)); }
  if (weight_) { cuda_assert(hipFree(weight_)); }
  if (action_list_) { cuda_assert(hipFree(action_list_)); }
}

ConcurrentGPUNeuralNet::EvaluationOrder ConcurrentGPUNeuralNet::compare_connections(const Connection& a, const Connection& b) {
    // A recurrent connection must be used before the origin is overwritten.
  if (a.type == ConnectionType::Recurrent && a.origin == b.dest) { return EvaluationOrder::LessThan; }
  if (b.type == ConnectionType::Recurrent && b.origin == a.dest) { return EvaluationOrder::GreaterThan; }

  // A normal connection must occur after every connection incoming to its origin has completed.
  if (a.type == ConnectionType::Normal && a.dest == b.origin) { return EvaluationOrder::LessThan; }
  if (b.type == ConnectionType::Normal && b.dest == a.origin) { return EvaluationOrder::GreaterThan; }

  // Two connections writing to the same destination must be in different sets.
  if (a.dest == b.dest) {
    // A self-recurrent connection happens at the same time as
    // zero-ing out, and so must occur first of all connections
    // writing to that node.
    if(a.origin == a.dest) {
      return EvaluationOrder::LessThan;
    } else if (b.origin == b.dest) {
      return EvaluationOrder::GreaterThan;
    } else {
      // Choice here is absolutely arbitrary.
      // This is arbitrary, and consistent.
      if(a.origin < b.origin) {
        return EvaluationOrder::GreaterThan;
      } else {
        return EvaluationOrder::LessThan;
      }
    }
  }

  // else a & b are not adjacent and cannot be compared
  return EvaluationOrder::Unknown;
}

void ConcurrentGPUNeuralNet::sort_connections(unsigned int first, unsigned int num_connections) {
  if(connections_sorted) {
    return;
  }

  // if the first connection in the list to sort is not
  // the first connection, and num_connections is zero
  // this is an error
  assert(!(first!=0 && num_connections==0));
  // if num_connections is zero, then we will sort all connections
  num_connections = num_connections > 0 ? num_connections : connections.size();
  // the number of connections to sort cannot be
  // larger than the total number of connections
  assert(first+num_connections <= connections.size());

  // zero out connection set index for use in sorting
  for (auto i=first; i<first+num_connections; i++) {
    connections[i].set = 0;
  }
  for(auto i=first; i<first+num_connections; i++) {
    for(auto j=i+1; j<first+num_connections; j++) {
      Connection& conn1 = connections[i];
      Connection& conn2 = connections[j];
      switch(compare_connections(conn1,conn2)) {
        case EvaluationOrder::GreaterThan:
          conn1.set++;
          break;

        case EvaluationOrder::LessThan:
          conn2.set++;
          break;

        case EvaluationOrder::Unknown:
          break;
      }
    }
  }

  auto split_iter = connections.begin()+first;
  auto last_iter = connections.begin()+first+num_connections;
  size_t current_set_num = 0;
  while(split_iter != last_iter) {
    auto next_split = std::partition(split_iter, last_iter,
                                     [](const Connection& conn) {
                                       return conn.set == 0;
                                     });
    assert(next_split != split_iter);

    // These could be run now, no longer need to track number of
    // depencencies.
    for(auto iter = split_iter; iter<next_split; iter++) {
      iter->set = current_set_num;
    }
    current_set_num++;

    // Decrease number of dependencies for everything else.
    for(auto iter_done = split_iter; iter_done<next_split; iter_done++) {
      for(auto iter_not_done = next_split; iter_not_done<last_iter; iter_not_done++) {
        if (compare_connections(*iter_done,*iter_not_done) == EvaluationOrder::LessThan) {
          iter_not_done->set--;
        }
      }
    }

    split_iter = next_split;
  }

  // build the action list if num_connections was the total set
  // or if this is the last subset of connections (all others are sorted)
  if (first + num_connections == connections.size()) {
    // if first is nonzero then we have been sorting based on subsets and now all subset lock free buckets
    // need to be merged in a sort of the entire connections list where set now is the lock free set index
    // (before it was used as the subnet index)

    if (first != 0) {
      // sort connections based on evaluation set number if not already done
      std::sort(connections.begin(),connections.end(),[](Connection a, Connection b){ return a.set < b.set; });
    }
    // build struct of arrays for use on GPU
    for (auto i=0u; i<connections.size(); i++) {
      auto& conn = connections[i];
      connection_list.add(conn.origin,conn.dest,conn.weight);
    }
    build_action_list();
    connections_sorted = true;
    connections.clear();
    synchronize();
  }
}

void ConcurrentGPUNeuralNet::ConcurrentGPUNeuralNet::build_action_list() {

  unsigned int num_connection_sets = connections.back().set+1;
  std::vector<unsigned int> connection_set_sizes(num_connection_sets, 0);
  for(auto& conn : connections) {
    connection_set_sizes[conn.set]++;
  }

  std::vector<unsigned int> earliest_zero_out_indices(nodes.size(), 0);
  std::vector<unsigned int> earliest_sigmoid_indices(nodes.size(), 0);

  std::vector<unsigned int> latest_zero_out_indices(nodes.size(), num_connection_sets);
  std::vector<unsigned int> latest_sigmoid_indices(nodes.size(), num_connection_sets);

  std::set<unsigned int> self_recurrent_nodes;

  for(auto& conn : connections) {
    // delay earliest possible zeroing of recurrent connections origins
    // until recurrent connections are applied
    if(conn.type == ConnectionType::Recurrent) {
      earliest_zero_out_indices[conn.origin] = std::max(
        earliest_zero_out_indices[conn.origin],
        conn.set + 1);
    }

    earliest_sigmoid_indices[conn.dest] = std::max(
      earliest_sigmoid_indices[conn.dest],
      conn.set + 1);

    latest_zero_out_indices[conn.dest] = std::min(
      latest_zero_out_indices[conn.dest],
      conn.set);

    if(conn.type == ConnectionType::Normal) {
      latest_sigmoid_indices[conn.origin] = std::min(
        latest_sigmoid_indices[conn.origin],
        conn.set);
    }

    if(conn.origin == conn.dest) {
      self_recurrent_nodes.insert(conn.origin);
    }
  }

  std::vector<unsigned int>& zero_out_indices = earliest_zero_out_indices;
  std::vector<unsigned int>& sigmoid_indices = earliest_sigmoid_indices;


  std::vector<std::vector<unsigned int> > zero_out_sets(num_connection_sets+1);
  std::vector<std::vector<unsigned int> > sigmoid_sets(num_connection_sets+1);

  for(unsigned int i=0; i<nodes.size(); i++) {
    bool is_self_recurrent = self_recurrent_nodes.count(i);
    if(!is_self_recurrent && i >= num_inputs) {
      zero_out_sets[zero_out_indices[i]].push_back(i);
    }
    if(i >= num_inputs) {
      sigmoid_sets[sigmoid_indices[i]].push_back(i);
    }
  }

  auto append_zero_out_set = [&](unsigned int i) {
    auto& zero_out_set = zero_out_sets[i];
    action_list.push_back(zero_out_set.size());
    for(unsigned int j : zero_out_set) {
      action_list.push_back(j);
    }
  };

  auto append_sigmoid_set = [&](unsigned int i) {
    auto& sigmoid_set = sigmoid_sets[i];
    action_list.push_back(sigmoid_set.size());
    for(unsigned int j : sigmoid_set) {
      action_list.push_back(j);
    }
  };



  action_list.clear();
  for(unsigned int i=0; i<num_connection_sets; i++) {
    append_zero_out_set(i);
    append_sigmoid_set(i);
    action_list.push_back(connection_set_sizes[i]);
  }

  append_zero_out_set(num_connection_sets);
  append_sigmoid_set(num_connection_sets);



  // print action list
  // for (auto& item : action_list) {
  //   std::cout << item << " ";
  // } std::cout << std::endl;
}


////////////////////////////////////////////////////////////////////////////

void ConcurrentGPUNeuralNet::add_node(NodeType type, ActivationFunction func) {
  switch (type) {
  case NodeType::Bias:
    num_inputs++;
    nodes.push_back(1.0);
    break;
  case NodeType::Input:
    num_inputs++;
    nodes.push_back(0.0);
    break;
  case NodeType::Output:
    num_outputs++;
    nodes.push_back(0.0);
    break;
  case NodeType::Hidden:
    nodes.push_back(0.0);
    break;
  }

  // Only sigmoid nodes implementated for ConcurrentGPUNeuralNet
  assert(func == ActivationFunction::Sigmoid);

}

_float_ sigmoid(_float_ val) {
  return 1/(1 + std::exp(-val));
}

void clear_nodes(unsigned int* list, _float_* nodes, unsigned int n) {
  for(auto i=0u; i<n; i++) {
    nodes[list[i]] = 0;
  }
}

void sigmoid_nodes(unsigned int* list, _float_* nodes, unsigned int n) {
  for(auto i=0u; i<n; i++) {
    nodes[list[i]] = sigmoid(nodes[list[i]]);
  }
}

void apply_connections(_float_* node, unsigned int* origin, unsigned int* dest, _float_* weight, unsigned int n) {
  for(auto i=0u; i<n; i++) {
    auto& conn_origin = origin[i];
    auto& conn_dest = dest[i];
    auto& conn_weight = weight[i];
    if(conn_origin == conn_dest) {
      // Special case for self-recurrent nodes
      // Be sure not to zero-out before this step.
      node[conn_origin] *= conn_weight;
    } else {
      node[conn_dest] += conn_weight*node[conn_origin];
    }
  }
}

__device__ _float_ device_sigmoid(_float_ val) {
  return 1/(1 + expf(-val));
}

__global__ void device_clear_nodes(unsigned int* list, _float_* nodes, unsigned int n) {
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  if (i<n) {
    nodes[list[i]] = 0;
  }
}

__global__ void device_sigmoid_nodes(unsigned int* list, _float_* nodes, unsigned int n) {
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  if (i<n) {
    nodes[list[i]] = device_sigmoid(nodes[list[i]]);
  }
}

__global__ void device_apply_connections(_float_* node, unsigned int* origin, unsigned int* dest, _float_* weight, unsigned int n) {
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  if (i<n) {
    auto& conn_origin = origin[i];
    auto& conn_dest = dest[i];
    auto& conn_weight = weight[i];
    if(conn_origin == conn_dest) {
      // Special case for self-recurrent nodes
      // Be sure not to zero-out before this step.
      node[conn_origin] *= conn_weight;
    } else {
      node[conn_dest] += conn_weight*node[conn_origin];
    }
  }
}

std::vector<_float_> ConcurrentGPUNeuralNet::host_evaluate(std::vector<_float_> inputs) {
  assert(inputs.size() == num_inputs-1);
  sort_connections();

  // copy inputs in to network
  std::copy(inputs.begin(),inputs.end(),nodes.begin()+1);

  auto i = 0u;
  int how_many_zero_out = action_list[i++];
  clear_nodes(&action_list[i], nodes.data(), how_many_zero_out);
  i += how_many_zero_out;

  int how_many_sigmoid = action_list[i++];
  sigmoid_nodes(&action_list[i], nodes.data(), how_many_sigmoid);
  i += how_many_sigmoid;

  int current_conn = 0;
  while(i<action_list.size()) {
    int how_many_conn = action_list[i++];
    apply_connections(nodes.data(), &connection_list.origin[current_conn], &connection_list.dest[current_conn], &connection_list.weight[current_conn], how_many_conn);
    current_conn += how_many_conn;

    int how_many_zero_out = action_list[i++];
    clear_nodes(&action_list[i], nodes.data(), how_many_zero_out);
    i += how_many_zero_out;

    int how_many_sigmoid = action_list[i++];
    sigmoid_nodes(&action_list[i], nodes.data(), how_many_sigmoid);
    i += how_many_sigmoid;
  }

  return std::vector<_float_> (nodes.begin()+num_inputs,nodes.begin()+num_inputs+num_outputs);
}

std::vector<_float_> ConcurrentGPUNeuralNet::evaluate(std::vector<_float_> inputs) {
  assert(inputs.size() == num_inputs-1);
  sort_connections();
  unsigned int num_blocks = 0;

  // copy inputs in to network
  //std::copy(inputs.begin(),inputs.end(),nodes.begin());
  cuda_assert(hipMemcpy(&node_[1],inputs.data(),inputs.size()*sizeof(_float_),hipMemcpyHostToDevice));

  auto i = 0u;
  int how_many_zero_out = action_list[i++];
  num_blocks = (how_many_zero_out+num_threads-1)/num_threads;
  if (how_many_zero_out) { device_clear_nodes<<<num_blocks,num_threads>>>(&action_list_[i], node_, how_many_zero_out); }
  i += how_many_zero_out;

  int how_many_sigmoid = action_list[i++];
  num_blocks = (how_many_sigmoid+num_threads-1)/num_threads;
  if (how_many_sigmoid) { device_sigmoid_nodes<<<num_blocks,num_threads>>>(&action_list_[i], node_, how_many_sigmoid); }
  i += how_many_sigmoid;

  int current_conn = 0;
  while(i<action_list.size()) {
    int how_many_conn = action_list[i++];
    num_blocks = (how_many_conn+num_threads-1)/num_threads;
    if (how_many_conn) { device_apply_connections<<<num_blocks,num_threads>>>(node_, &origin_[current_conn], &dest_[current_conn], &weight_[current_conn], how_many_conn); }
    current_conn += how_many_conn;

    int how_many_zero_out = action_list[i++];
    num_blocks = (how_many_zero_out+num_threads-1)/num_threads;
    if (how_many_zero_out) { device_clear_nodes<<<num_blocks,num_threads>>>(&action_list_[i], node_, how_many_zero_out); }
    i += how_many_zero_out;

    int how_many_sigmoid = action_list[i++];
    num_blocks = (how_many_sigmoid+num_threads-1)/num_threads;
    if (how_many_sigmoid) { device_sigmoid_nodes<<<num_blocks,num_threads>>>(&action_list_[i], node_, how_many_sigmoid); }
    i += how_many_sigmoid;
  }
  cuda_assert(hipDeviceSynchronize());
  std::vector<_float_> outputs(num_outputs,0);
  cuda_assert(hipMemcpy(outputs.data(),&node_[num_inputs],num_outputs*sizeof(_float_),hipMemcpyDeviceToHost));

  return outputs;
}

void ConcurrentGPUNeuralNet::add_connection(int origin, int dest, _float_ weight, unsigned int set) {
  if(would_make_loop(origin,dest,set)) {
    connections.emplace_back(origin,dest,ConnectionType::Recurrent,weight,set);
  } else {
    connections.emplace_back(origin,dest,ConnectionType::Normal,weight,set);
  }
}

bool ConcurrentGPUNeuralNet::would_make_loop(unsigned int i, unsigned int j, unsigned int set) {
  // handle the case of a recurrent connection to itself up front
  if (i == j) { return true; }

  if (set == std::numeric_limits<unsigned int>::max()) {

    std::vector<bool> reachable(nodes.size(), false);
    reachable[j] = true;

    while (true) {

      bool found_new_node = false;
      for (auto const& conn : connections) {
        // if the origin of this connection is reachable and its
        // desitination is not, then it should be made reachable
        if (reachable[conn.origin] &&
                    !reachable[conn.dest] &&
            conn.type == ConnectionType::Normal) {
          // if it is a normal node. if it is the origin of the
          // proposed additional connection (i->j) then it would be
          // a loop
          if (conn.dest == i) {
            // the destination of this reachable connection is
            // the origin of the proposed connection. thus there
            // exists a path from j -> i. So this will be a loop.
            return true;
          }
          else {
            reachable[conn.dest] = true;
            found_new_node = true;
          }
        }
      }
      // no loop detected
      if (!found_new_node) {
        return false;
      }

    }

  } else {
    // if set number is not zero, then it is assumed the added connection is
    // part of a subnet that is currently being added.

    std::map<unsigned int,unsigned int> subset_node_map;
    subset_node_map[i] = subset_node_map.size();
    subset_node_map[j] = subset_node_map.size();

    auto conn_iter = connections.end();
    while (conn_iter-- != connections.begin()) {
      auto conn_set = (*conn_iter).set;
      if (conn_set != set){
        break;
      } else {
        auto origin = (*conn_iter).origin;
        auto dest = (*conn_iter).dest;

        if (subset_node_map.count(origin)==0) {
          subset_node_map[origin] = subset_node_map.size();
        }
        if (subset_node_map.count(dest)==0) {
          subset_node_map[dest] = subset_node_map.size();
        }
      }
    }

    std::vector<bool> reachable(subset_node_map.size(), false);
    reachable[subset_node_map[j]] = true;

    while (true) {
      auto conn_start = conn_iter;

      bool found_new_node = false;
      while (++conn_start != connections.end()) {
        auto const& conn = *conn_start;
        assert(conn.set == set);

        // if the origin of this connection is reachable and its
        // desitination is not, then it should be made reachable
        if (reachable[subset_node_map[conn.origin]] &&
            !reachable[subset_node_map[conn.dest]] &&
            conn.type == ConnectionType::Normal) {
          // if it is a normal node. if it is the origin of the
          // proposed additional connection (i->j) then it would be
          // a loop
          if (conn.dest == i) {
            // the destination of this reachable connection is
            // the origin of the proposed connection. thus there
            // exists a path from j -> i. So this will be a loop.
            return true;
          }
          else {
            reachable[subset_node_map[conn.dest]] = true;
            found_new_node = true;
          }
        }
      }
      // no loop detected
      if (!found_new_node) {
        return false;
      }

    }
  }
}

// TODO: implement gpu_smart_pointer to handle GPU memory according to RAII
void ConcurrentGPUNeuralNet::synchronize() {
  cuda_assert(hipMalloc((void**)&node_,nodes.size()*sizeof(_float_)));
  cuda_assert(hipMemcpy(node_,nodes.data(),nodes.size()*sizeof(_float_),hipMemcpyHostToDevice));

  cuda_assert(hipMalloc((void**)&origin_,connection_list.origin.size()*sizeof(unsigned int)));
  cuda_assert(hipMemcpy(origin_,connection_list.origin.data(),connection_list.origin.size()*sizeof(unsigned int),hipMemcpyHostToDevice));

  cuda_assert(hipMalloc((void**)&dest_,connection_list.dest.size()*sizeof(unsigned int)));
  cuda_assert(hipMemcpy(dest_,connection_list.dest.data(),connection_list.dest.size()*sizeof(unsigned int),hipMemcpyHostToDevice));
  
  cuda_assert(hipMalloc((void**)&weight_,connection_list.weight.size()*sizeof(_float_)));
  cuda_assert(hipMemcpy(weight_,connection_list.weight.data(),connection_list.weight.size()*sizeof(_float_),hipMemcpyHostToDevice));

  cuda_assert(hipMalloc((void**)&action_list_,action_list.size()*sizeof(unsigned int)));
  cuda_assert(hipMemcpy(action_list_,action_list.data(),action_list.size()*sizeof(unsigned int),hipMemcpyHostToDevice));
}


void ConcurrentGPUNeuralNet::print_network(std::ostream& os) const {
  std::stringstream ss; ss.str("");
  ss << "Action List: \n\n";

  auto i = 0u;
  int how_many_zero_out = action_list[i++];
  ss << "# Zero out: " << how_many_zero_out << "\n";
  i += how_many_zero_out;

  int how_many_sigmoid = action_list[i++];
  ss << "# Sigmoid: " << how_many_sigmoid << "\n";
  i += how_many_sigmoid;

  std::vector<unsigned int> num_conn_to_apply;
  int current_conn = 0;
  while(i<action_list.size()) {
    int how_many_conn = action_list[i++];
    ss << "# Connections: " << how_many_conn << "\n";
    current_conn += how_many_conn;
    num_conn_to_apply.push_back(how_many_conn);

    int how_many_zero_out = action_list[i++];
    ss << "# Zero out: " << how_many_zero_out << "\n";
    i += how_many_zero_out;

    int how_many_sigmoid = action_list[i++];
    ss << "# Sigmoid: " << how_many_sigmoid << "\n";
    i += how_many_sigmoid;
  }
  os << ss.str();

  // ss.str("");
  // ss << "\nConnection sets:\n";
  // int counter = 0;
  // int num = num_conn_to_apply[counter];
  // for (auto i=0u; i<connection_list.size(); i++) {
  //   ss << connection_list.origin[i] << " -> " << connection_list.dest[i] << "\n";
  //   if (i == num-1) { num += num_conn_to_apply[++counter]; ss << "\n";}
  // }

  // os << ss.str();
}
