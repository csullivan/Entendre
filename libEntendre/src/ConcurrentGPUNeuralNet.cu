#include "hip/hip_runtime.h"
#include "ConcurrentGPUNeuralNet.hh"

#include <cassert>
#include <stdexcept>
#include <cmath>
#include <iostream>
#include <map>
#include <set>
#include <sstream>
#include <vector>
#include <algorithm>

#include "logging.h"
#include "math.h"


#define cuda_assert(ans) { cudaAssert((ans), __FILE__, __LINE__); }

inline void cudaAssert(hipError_t code, const char *file, int line, bool abort=true) {
//#ifndef NDEBUG
  if (code != hipSuccess) {
    fprintf(stderr,"cuda_assert: %s %s %d\n", hipGetErrorString(code), file, line);
    if (abort) { throw code; }
  }
//#endif
}

ConcurrentGPUNeuralNet::~ConcurrentGPUNeuralNet() {
  if (node_) { cuda_assert(hipFree(node_)); }
  if (origin_) { cuda_assert(hipFree(origin_)); }
  if (dest_) { cuda_assert(hipFree(dest_)); }
  if (weight_) { cuda_assert(hipFree(weight_)); }
  if (action_list_) { cuda_assert(hipFree(action_list_)); }
}

ConcurrentGPUNeuralNet::EvaluationOrder ConcurrentGPUNeuralNet::compare_connections(const Connection& a, const Connection& b) {
    // A recurrent connection must be used before the origin is overwritten.
  if (a.type == ConnectionType::Recurrent && a.origin == b.dest) { return EvaluationOrder::LessThan; }
  if (b.type == ConnectionType::Recurrent && b.origin == a.dest) { return EvaluationOrder::GreaterThan; }

  // A normal connection must occur after every connection incoming to its origin has completed.
  if (a.type == ConnectionType::Normal && a.dest == b.origin) { return EvaluationOrder::LessThan; }
  if (b.type == ConnectionType::Normal && b.dest == a.origin) { return EvaluationOrder::GreaterThan; }

  // Two connections writing to the same destination must be in different sets.
  if (a.dest == b.dest) {
    // A self-recurrent connection happens at the same time as
    // zero-ing out, and so must occur first of all connections
    // writing to that node.
    if(a.origin == a.dest) {
      return EvaluationOrder::LessThan;
    } else if (b.origin == b.dest) {
      return EvaluationOrder::GreaterThan;
    } else {
      // Choice here is absolutely arbitrary.
      // This is arbitrary, and consistent.
      if(a.origin < b.origin) {
        return EvaluationOrder::GreaterThan;
      } else {
        return EvaluationOrder::LessThan;
      }
    }
  }

  // else a & b are not adjacent and cannot be compared
  return EvaluationOrder::Unknown;
}

void ConcurrentGPUNeuralNet::sort_connections(unsigned int first, unsigned int num_connections) {
  if(connections_sorted) {
    return;
  }

  // if the first connection in the list to sort is not
  // the first connection, and num_connections is zero
  // this is an error
  assert(!(first!=0 && num_connections==0));
  // if num_connections is zero, then we will sort all connections
  num_connections = num_connections > 0 ? num_connections : connections.size();
  // the number of connections to sort cannot be
  // larger than the total number of connections
  assert(first+num_connections <= connections.size());

  // zero out connection set index for use in sorting
  for (auto i=first; i<first+num_connections; i++) {
    connections[i].set = 0;
  }
  for(auto i=first; i<first+num_connections; i++) {
    for(auto j=i+1; j<first+num_connections; j++) {
      Connection& conn1 = connections[i];
      Connection& conn2 = connections[j];
      switch(compare_connections(conn1,conn2)) {
        case EvaluationOrder::GreaterThan:
          conn1.set++;
          break;

        case EvaluationOrder::LessThan:
          conn2.set++;
          break;

        case EvaluationOrder::Unknown:
          break;
      }
    }
  }

  auto split_iter = connections.begin()+first;
  auto last_iter = connections.begin()+first+num_connections;
  size_t current_set_num = 0;
  while(split_iter != last_iter) {
    auto next_split = std::partition(split_iter, last_iter,
                                     [](const Connection& conn) {
                                       return conn.set == 0;
                                     });
    assert(next_split != split_iter);

    // These could be run now, no longer need to track number of
    // depencencies.
    for(auto iter = split_iter; iter<next_split; iter++) {
      iter->set = current_set_num;
    }
    current_set_num++;

    // Decrease number of dependencies for everything else.
    for(auto iter_done = split_iter; iter_done<next_split; iter_done++) {
      for(auto iter_not_done = next_split; iter_not_done<last_iter; iter_not_done++) {
        if (compare_connections(*iter_done,*iter_not_done) == EvaluationOrder::LessThan) {
          iter_not_done->set--;
        }
      }
    }

    split_iter = next_split;
  }

  // build the action list if num_connections was the total set
  // or if this is the last subset of connections (all others are sorted)
  if (first + num_connections == connections.size()) {
    // if first is nonzero then we have been sorting based on subsets and now all subset lock free buckets
    // need to be merged in a sort of the entire connections list where set now is the lock free set index
    // (before it was used as the subnet index)

    if (first != 0) {
      // sort connections based on evaluation set number if not already done
      std::sort(connections.begin(),connections.end(),[](Connection a, Connection b){ return a.set < b.set; });
    }
    // build struct of arrays for use on GPU
    for (auto i=0u; i<connections.size(); i++) {
      auto& conn = connections[i];
      connection_list.add(conn.origin,conn.dest,conn.weight);
    }
    build_action_list();
    connections_sorted = true;
    connections.clear();
    synchronize();
  }
}

void ConcurrentGPUNeuralNet::ConcurrentGPUNeuralNet::build_action_list() {

  unsigned int num_connection_sets = connections.back().set+1;
  std::vector<unsigned int> connection_set_sizes(num_connection_sets, 0);
  for(auto& conn : connections) {
    connection_set_sizes[conn.set]++;
  }

  std::vector<unsigned int> earliest_zero_out_indices(nodes.size(), 0);
  std::vector<unsigned int> earliest_sigmoid_indices(nodes.size(), 0);

  std::vector<unsigned int> latest_zero_out_indices(nodes.size(), num_connection_sets);
  std::vector<unsigned int> latest_sigmoid_indices(nodes.size(), num_connection_sets);

  std::set<unsigned int> self_recurrent_nodes;

  for(auto& conn : connections) {
    // delay earliest possible zeroing of recurrent connections origins
    // until recurrent connections are applied
    if(conn.type == ConnectionType::Recurrent) {
      earliest_zero_out_indices[conn.origin] = std::max(
        earliest_zero_out_indices[conn.origin],
        conn.set + 1);
    }

    earliest_sigmoid_indices[conn.dest] = std::max(
      earliest_sigmoid_indices[conn.dest],
      conn.set + 1);

    latest_zero_out_indices[conn.dest] = std::min(
      latest_zero_out_indices[conn.dest],
      conn.set);

    if(conn.type == ConnectionType::Normal) {
      latest_sigmoid_indices[conn.origin] = std::min(
        latest_sigmoid_indices[conn.origin],
        conn.set);
    }

    if(conn.origin == conn.dest) {
      self_recurrent_nodes.insert(conn.origin);
    }
  }

  std::vector<unsigned int>& zero_out_indices = earliest_zero_out_indices;
  std::vector<unsigned int>& sigmoid_indices = earliest_sigmoid_indices;


  std::vector<std::vector<unsigned int> > zero_out_sets(num_connection_sets+1);
  std::vector<std::vector<unsigned int> > sigmoid_sets(num_connection_sets+1);

  for(unsigned int i=0; i<nodes.size(); i++) {
    bool is_self_recurrent = self_recurrent_nodes.count(i);
    if(!is_self_recurrent && i >= num_inputs) {
      zero_out_sets[zero_out_indices[i]].push_back(i);
    }
    if(i >= num_inputs) {
      sigmoid_sets[sigmoid_indices[i]].push_back(i);
    }
  }

  auto append_zero_out_set = [&](unsigned int i) {
    auto& zero_out_set = zero_out_sets[i];
    action_list.push_back(zero_out_set.size());
    for(unsigned int j : zero_out_set) {
      action_list.push_back(j);
    }
  };

  auto append_sigmoid_set = [&](unsigned int i) {
    auto& sigmoid_set = sigmoid_sets[i];
    action_list.push_back(sigmoid_set.size());
    for(unsigned int j : sigmoid_set) {
      action_list.push_back(j);
    }
  };



  action_list.clear();
  for(unsigned int i=0; i<num_connection_sets; i++) {
    append_zero_out_set(i);
    append_sigmoid_set(i);
    action_list.push_back(connection_set_sizes[i]);
  }

  append_zero_out_set(num_connection_sets);
  append_sigmoid_set(num_connection_sets);



  // print action list
  // for (auto& item : action_list) {
  //   std::cout << item << " ";
  // } std::cout << std::endl;
}


////////////////////////////////////////////////////////////////////////////

void ConcurrentGPUNeuralNet::add_node(const NodeType& type) {
switch (type) {
  case NodeType::Bias:
    num_inputs++;
    nodes.push_back(1.0);
    break;
  case NodeType::Input:
    num_inputs++;
    nodes.push_back(0.0);
    break;
  case NodeType::Output:
    num_outputs++;
    nodes.push_back(0.0);
    break;
  case NodeType::Hidden:
    nodes.push_back(0.0);
    break;
  };
}

_float_ sigmoid(_float_ val) {
  return 1/(1 + std::exp(-val));
}

void clear_nodes(unsigned int* list, _float_* nodes, unsigned int n) {
  for(auto i=0u; i<n; i++) {
    nodes[list[i]] = 0;
  }
}

void sigmoid_nodes(unsigned int* list, _float_* nodes, unsigned int n) {
  for(auto i=0u; i<n; i++) {
    nodes[list[i]] = sigmoid(nodes[list[i]]);
  }
}

void apply_connections(_float_* node, unsigned int* origin, unsigned int* dest, _float_* weight, unsigned int n) {
  for(auto i=0u; i<n; i++) {
    auto& conn_origin = origin[i];
    auto& conn_dest = dest[i];
    auto& conn_weight = weight[i];
    if(conn_origin == conn_dest) {
      // Special case for self-recurrent nodes
      // Be sure not to zero-out before this step.
      node[conn_origin] *= conn_weight;
    } else {
      node[conn_dest] += conn_weight*node[conn_origin];
    }
  }
}

__device__ _float_ device_sigmoid(_float_ val) {
  return 1/(1 + expf(-val));
}

__global__ void device_clear_nodes(unsigned int* list, _float_* nodes, unsigned int n) {
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  if (i<n) {
    nodes[list[i]] = 0;
  }
}

__global__ void device_sigmoid_nodes(unsigned int* list, _float_* nodes, unsigned int n) {
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  if (i<n) {
    nodes[list[i]] = device_sigmoid(nodes[list[i]]);
  }
}

__global__ void device_apply_connections(_float_* node, unsigned int* origin, unsigned int* dest, _float_* weight, unsigned int n) {
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  if (i<n) {
    auto& conn_origin = origin[i];
    auto& conn_dest = dest[i];
    auto& conn_weight = weight[i];
    if(conn_origin == conn_dest) {
      // Special case for self-recurrent nodes
      // Be sure not to zero-out before this step.
      node[conn_origin] *= conn_weight;
    } else {
      node[conn_dest] += conn_weight*node[conn_origin];
    }
  }
}

std::vector<_float_> ConcurrentGPUNeuralNet::host_evaluate(std::vector<_float_> inputs) {
  assert(inputs.size() == num_inputs-1);
  sort_connections();

  // copy inputs in to network
  std::copy(inputs.begin(),inputs.end(),nodes.begin()+1);

  auto i = 0u;
  int how_many_zero_out = action_list[i++];
  clear_nodes(&action_list[i], nodes.data(), how_many_zero_out);
  i += how_many_zero_out;

  int how_many_sigmoid = action_list[i++];
  sigmoid_nodes(&action_list[i], nodes.data(), how_many_sigmoid);
  i += how_many_sigmoid;

  int current_conn = 0;
  while(i<action_list.size()) {
    int how_many_conn = action_list[i++];
    apply_connections(nodes.data(), &connection_list.origin[current_conn], &connection_list.dest[current_conn], &connection_list.weight[current_conn], how_many_conn);
    current_conn += how_many_conn;

    int how_many_zero_out = action_list[i++];
    clear_nodes(&action_list[i], nodes.data(), how_many_zero_out);
    i += how_many_zero_out;

    int how_many_sigmoid = action_list[i++];
    sigmoid_nodes(&action_list[i], nodes.data(), how_many_sigmoid);
    i += how_many_sigmoid;
  }

  return std::vector<_float_> (nodes.begin()+num_inputs,nodes.begin()+num_inputs+num_outputs);
}

std::vector<_float_> ConcurrentGPUNeuralNet::evaluate(std::vector<_float_> inputs) {
  assert(inputs.size() == num_inputs-1);
  sort_connections();
  unsigned int num_blocks = 0;

  // copy inputs in to network
  //std::copy(inputs.begin(),inputs.end(),nodes.begin());
  cuda_assert(hipMemcpy(&node_[1],inputs.data(),inputs.size()*sizeof(_float_),hipMemcpyHostToDevice));

  auto i = 0u;
  int how_many_zero_out = action_list[i++];
  num_blocks = (how_many_zero_out+num_threads-1)/num_threads;
  if (how_many_zero_out) { device_clear_nodes<<<num_blocks,num_threads>>>(&action_list_[i], node_, how_many_zero_out); }
  i += how_many_zero_out;

  int how_many_sigmoid = action_list[i++];
  num_blocks = (how_many_sigmoid+num_threads-1)/num_threads;
  if (how_many_sigmoid) { device_sigmoid_nodes<<<num_blocks,num_threads>>>(&action_list_[i], node_, how_many_sigmoid); }
  i += how_many_sigmoid;

  int current_conn = 0;
  while(i<action_list.size()) {
    int how_many_conn = action_list[i++];
    num_blocks = (how_many_conn+num_threads-1)/num_threads;
    if (how_many_conn) { device_apply_connections<<<num_blocks,num_threads>>>(node_, &origin_[current_conn], &dest_[current_conn], &weight_[current_conn], how_many_conn); }
    current_conn += how_many_conn;

    int how_many_zero_out = action_list[i++];
    num_blocks = (how_many_zero_out+num_threads-1)/num_threads;
    if (how_many_zero_out) { device_clear_nodes<<<num_blocks,num_threads>>>(&action_list_[i], node_, how_many_zero_out); }
    i += how_many_zero_out;

    int how_many_sigmoid = action_list[i++];
    num_blocks = (how_many_sigmoid+num_threads-1)/num_threads;
    if (how_many_sigmoid) { device_sigmoid_nodes<<<num_blocks,num_threads>>>(&action_list_[i], node_, how_many_sigmoid); }
    i += how_many_sigmoid;
  }
  cuda_assert(hipDeviceSynchronize());
  std::vector<_float_> outputs(num_outputs,0);
  cuda_assert(hipMemcpy(outputs.data(),&node_[num_inputs],num_outputs*sizeof(_float_),hipMemcpyDeviceToHost));

  return outputs;
}

void ConcurrentGPUNeuralNet::add_connection(int origin, int dest, _float_ weight, unsigned int set) {
  if(would_make_loop(origin,dest,set)) {
    connections.emplace_back(origin,dest,ConnectionType::Recurrent,weight,set);
  } else {
    connections.emplace_back(origin,dest,ConnectionType::Normal,weight,set);
  }
}

bool ConcurrentGPUNeuralNet::would_make_loop(unsigned int i, unsigned int j, unsigned int set) {
  // handle the case of a recurrent connection to itself up front
  if (i == j) { return true; }

  if (set == std::numeric_limits<unsigned int>::max()) {

    std::vector<bool> reachable(nodes.size(), false);
    reachable[j] = true;

    while (true) {

      bool found_new_node = false;
      for (auto const& conn : connections) {
        // if the origin of this connection is reachable and its
        // desitination is not, then it should be made reachable
        if (reachable[conn.origin] &&
                    !reachable[conn.dest] &&
            conn.type == ConnectionType::Normal) {
          // if it is a normal node. if it is the origin of the
          // proposed additional connection (i->j) then it would be
          // a loop
          if (conn.dest == i) {
            // the destination of this reachable connection is
            // the origin of the proposed connection. thus there
            // exists a path from j -> i. So this will be a loop.
            return true;
          }
          else {
            reachable[conn.dest] = true;
            found_new_node = true;
          }
        }
      }
      // no loop detected
      if (!found_new_node) {
        return false;
      }

    }

  } else {
    // if set number is not zero, then it is assumed the added connection is
    // part of a subnet that is currently being added.

    std::map<unsigned int,unsigned int> subset_node_map;
    subset_node_map[i] = subset_node_map.size();
    subset_node_map[j] = subset_node_map.size();

    auto conn_iter = connections.end();
    while (conn_iter-- != connections.begin()) {
      auto conn_set = (*conn_iter).set;
      if (conn_set != set){
        break;
      } else {
        auto origin = (*conn_iter).origin;
        auto dest = (*conn_iter).dest;

        if (subset_node_map.count(origin)==0) {
          subset_node_map[origin] = subset_node_map.size();
        }
        if (subset_node_map.count(dest)==0) {
          subset_node_map[dest] = subset_node_map.size();
        }
      }

    }


    std::vector<bool> reachable(subset_node_map.size(), false);
    reachable[subset_node_map[j]] = true;
    while (true) {
      auto conn_start = conn_iter;

      bool found_new_node = false;
      while (++conn_start != connections.end()) {
        auto const& conn = *conn_start;
        assert(conn.set == set);

        // if the origin of this connection is reachable and its
        // desitination is not, then it should be made reachable
        if (reachable[subset_node_map[conn.origin]] &&
            !reachable[subset_node_map[conn.dest]] &&
            conn.type == ConnectionType::Normal) {
          // if it is a normal node. if it is the origin of the
          // proposed additional connection (i->j) then it would be
          // a loop
          if (conn.dest == i) {
            // the destination of this reachable connection is
            // the origin of the proposed connection. thus there
            // exists a path from j -> i. So this will be a loop.
            return true;
          }
          else {
            reachable[subset_node_map[conn.dest]] = true;
            found_new_node = true;
          }
        }
      }
      // no loop detected
      if (!found_new_node) {
        return false;
      }

    }
  }
}

// TODO: implement gpu_smart_pointer to handle GPU memory according to RAII
void ConcurrentGPUNeuralNet::synchronize() {
  cuda_assert(hipMalloc((void**)&node_,nodes.size()*sizeof(_float_)));
  cuda_assert(hipMemcpy(node_,nodes.data(),nodes.size()*sizeof(_float_),hipMemcpyHostToDevice));

  cuda_assert(hipMalloc((void**)&origin_,connection_list.origin.size()*sizeof(unsigned int)));
  cuda_assert(hipMemcpy(origin_,connection_list.origin.data(),connection_list.origin.size()*sizeof(unsigned int),hipMemcpyHostToDevice));

  cuda_assert(hipMalloc((void**)&dest_,connection_list.dest.size()*sizeof(unsigned int)));
  cuda_assert(hipMemcpy(dest_,connection_list.dest.data(),connection_list.dest.size()*sizeof(unsigned int),hipMemcpyHostToDevice));
  
  cuda_assert(hipMalloc((void**)&weight_,connection_list.weight.size()*sizeof(_float_)));
  cuda_assert(hipMemcpy(weight_,connection_list.weight.data(),connection_list.weight.size()*sizeof(_float_),hipMemcpyHostToDevice));

  cuda_assert(hipMalloc((void**)&action_list_,action_list.size()*sizeof(unsigned int)));
  cuda_assert(hipMemcpy(action_list_,action_list.data(),action_list.size()*sizeof(unsigned int),hipMemcpyHostToDevice));
}


void ConcurrentGPUNeuralNet::print_network(std::ostream& os) const {
  std::stringstream ss; ss.str("");
  ss << "Action List: \n\n";

  auto i = 0u;
  int how_many_zero_out = action_list[i++];
  ss << "# Zero out: " << how_many_zero_out << "\n";
  i += how_many_zero_out;

  int how_many_sigmoid = action_list[i++];
  ss << "# Sigmoid: " << how_many_sigmoid << "\n";
  i += how_many_sigmoid;

  std::vector<unsigned int> num_conn_to_apply;
  int current_conn = 0;
  while(i<action_list.size()) {
    int how_many_conn = action_list[i++];
    ss << "# Connections: " << how_many_conn << "\n";
    current_conn += how_many_conn;
    num_conn_to_apply.push_back(how_many_conn);

    int how_many_zero_out = action_list[i++];
    ss << "# Zero out: " << how_many_zero_out << "\n";
    i += how_many_zero_out;

    int how_many_sigmoid = action_list[i++];
    ss << "# Sigmoid: " << how_many_sigmoid << "\n";
    i += how_many_sigmoid;
  }
  os << ss.str();

  // ss.str("");
  // ss << "\nConnection sets:\n";
  // int counter = 0;
  // int num = num_conn_to_apply[counter];
  // for (auto i=0u; i<connection_list.size(); i++) {
  //   ss << connection_list.origin[i] << " -> " << connection_list.dest[i] << "\n";
  //   if (i == num-1) { num += num_conn_to_apply[++counter]; ss << "\n";}
  // }

  // os << ss.str();
}